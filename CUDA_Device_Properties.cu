// CUDA_Device_Properties.cpp : Definiert den Einstiegspunkt f�r die Konsolenanwendung.
/*
* Description:
*	Acquire info about CUDA devices on system.
*
*	Author:	P Stegmann
*	Date: 2014-10-22
*/
//

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""


int main()
{
	// Get number of devices
	int nDevices = 0;
	hipError_t errorcode = hipSuccess;
	errorcode = hipGetDeviceCount(&nDevices);
	std::cout << "Anzahl der CUDA GPUs: " << nDevices << std::endl;
	std::cout << hipGetErrorString(errorcode) << std::endl;
	// Loop over all devices
	
	for(int ii = 0; ii < nDevices; ii++)
	{	
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, ii);
		printf("Device number: %d\n", ii);
		printf(" Device name: %s\n", prop.name);
		printf(" Minor Compute capability: %d\n", prop.minor);
		printf(" Major compute capability: %d\n", prop.major);
		printf(" Memory Clock Rate (KHz): %d\n", prop.memoryBusWidth);
		printf(" Peak Memory Bandwidth (GB/s) : %f\n", 
			2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		printf(" Maximum threads per block: %d\n", prop.maxThreadsPerBlock);
		printf(" Size of a warp: %d\n", prop.warpSize);
		printf(" Available shared memory: %d\n", prop.sharedMemPerBlock);
		printf(" Maximum global memory: %d\n", prop.totalGlobalMem);
	}
	
	return 0;
}

